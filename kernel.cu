﻿
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void vectorAdd(int* a, int* b, int* c)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
    
    return;
}

int main()
{
    int a[] = { 1, 2, 3 };
    int b[] = { 4, 5, 6 };
    int c[sizeof(a) / sizeof(int)] = { 0 };

    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;

    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);
    hipMemcpy(cudaC, c, sizeof(c), hipMemcpyHostToDevice);

    vectorAdd <<< 1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);
    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

    for (int i = 0; i < sizeof(c) / sizeof(int); i++)
    {
        c[i] = a[i] + b[i];
    }

    for (int i = 0; i < sizeof(c) / sizeof(int); i++)
    {
        printf("%d: %d\n", i, c[i]);
    }

    return 0;
}
